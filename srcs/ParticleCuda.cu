#include "ParticleCuda.cuh"


ParticleCuda::ParticleCuda() {}

ParticleCuda::~ParticleCuda() {
    hipFree(this->cu_position);
    hipFree(this->cu_velocity);
}

void ParticleCuda::initialize(const std::vector<glm::vec3> &position, const std::vector<glm::vec3> &velocity,
    const int particle_num, const int threads, const float collision_distance) {
    this->threads = threads;
    this->blocks = (particle_num + threads - 1) / threads;
    this->collision_distance = collision_distance;

    // Allocate device memory
    hipMalloc(&this->cu_position, particle_num * sizeof(glm::vec3));
    hipMalloc(&this->cu_velocity, particle_num * sizeof(glm::vec3));

    hipMemcpy(this->cu_position, position.data(), particle_num * sizeof(glm::vec3),
               hipMemcpyHostToDevice);
    hipMemcpy(this->cu_velocity, velocity.data(), particle_num * sizeof(glm::vec3),
               hipMemcpyHostToDevice);
}

void ParticleCuda::update_position_velocity(std::vector<glm::vec3> &position,
    const float mass, const float delta_time) {
    update_particle_kernel<<<this->blocks, this->threads>>>(
        this->cu_position, this->cu_velocity, mass, delta_time,
        position.size(), this->collision_distance);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        ParticleCuda::~ParticleCuda();
        exit(1);
    }
    hipDeviceSynchronize();

    hipMemcpy(position.data(), this->cu_position, position.size() * sizeof(glm::vec3),
               hipMemcpyDeviceToHost);
}
