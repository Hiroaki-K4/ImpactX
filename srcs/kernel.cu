#include "hip/hip_runtime.h"
#include "kernel.cuh"


__global__ void update_particle_kernel(glm::vec3 *cu_position, glm::vec3 *cu_velocity, float mass,
                                float delta_time, int num_particles, float collision_distance) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= num_particles) {
        return;
    }

    glm::vec3 all_accel(0.0f);
    for (int j = 0; j < num_particles; j++) {
        if (i == j) {
            continue;
        }
        float dist = glm::distance(cu_position[i], cu_position[j]);
        if (dist <= collision_distance) {
            // Calculate collision
            // TODO: Fix collision calculation
            if (i < j) {
                glm::vec3 temp = cu_velocity[i];
                cu_velocity[i] = cu_velocity[j];
                cu_velocity[j] = temp;
            }
        } else {
            // Calculate gravity
            float G = 6.67430e-11;
            float accel_power = G * (mass * mass) / (dist * dist);
            glm::vec3 accel = (cu_position[j] - cu_position[i]) / dist * accel_power;
            all_accel += accel;
        }
    }
    cu_velocity[i] += all_accel * delta_time;
    cu_position[i] += cu_velocity[i] * delta_time;
}
